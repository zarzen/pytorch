
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCStorage.cu"
#else

void THCStorage_(fill)(THCState *state, THCStorage *self, scalar_t value)
{
  at::cuda::ThrustAllocator thrustAlloc;
  thrust::device_ptr<scalar_t> self_data(THCStorage_(data)(state, self));
  thrust::fill(
#if (defined(CUDA_VERSION) && CUDA_VERSION >= 7000) || defined(USE_ROCM)
      thrust::cuda::par(thrustAlloc).on(c10::cuda::getCurrentCUDAStream()),
#endif
      self_data,
      self_data + (self->nbytes() / sizeof(scalar_t)),
      value);
}

void THCStorage_(
    resizeBytes)(THCState* state, THCStorage* self, ptrdiff_t size_bytes) {
  THCStorage_resizeBytes(state, self, size_bytes);
}

int THCStorage_(getDevice)(THCState* state, const THCStorage* storage) {
  return THCStorage_getDevice(state, storage);
}

#endif
